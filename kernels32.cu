
#include <hip/hip_runtime.h>
extern "C" __global__
void divElements(float * x, float * y, int n) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid < n) {
		x[tid] /= y[tid];
	}
}

extern "C" __global__
void elemMax(float * dst, float * src, int n) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid < n) {
		dst[tid] = max(dst[tid], src[tid]);
	}
}

extern "C" __global__
void expElements(float * x, int n) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid < n) {
		x[tid] = expf(x[tid]);
	}
}

extern "C" __global__
void logElements(float * x, int n) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid < n) {
		x[tid] = logf(x[tid]);
	}
}

extern "C" __global__
void tanhElements(float * x, int n) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid < n) {
		x[tid] = tanhf(x[tid]);
	}
}

extern "C" __global__
void sinElements(float * x, int n) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid < n) {
		x[tid] = sinf(x[tid]);
	}
}

extern "C" __global__
void sigmoidElements(float * x, int n) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid < n) {
		x[tid] = (1 + tanhf(x[tid] / 2)) / 2;
	}
}

extern "C" __global__
void clipPositive(float * x, int n) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid < n) {
		x[tid] = fmaxf(0, x[tid]);
	}
}

extern "C" __global__
void shiftRandUniform(float * x, int n) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid < n) {
		if (x[tid] == 1.0f) {
			x[tid] = 0;
		}
	}
}

extern "C" __global__
void uniformToBernoulli(float * x, int n) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid < n) {
		if (x[tid] > 0.5) {
			x[tid] = 1;
		} else {
			x[tid] = 0;
		}
	}
}

extern "C" __global__
void addRepeated(float * dest, float * source, int destLen, int sourceLen) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid < destLen) {
		dest[tid] += source[tid % sourceLen];
	}
}

extern "C" __global__
void addRepeatedPow2(float * dest, float * source, int destLen, int srcMask) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid < destLen) {
		dest[tid] += source[tid & srcMask];
	}
}

extern "C" __global__
void scaleRepeated(float * dest, float * source, int destLen, int sourceLen) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid < destLen) {
		dest[tid] *= source[tid % sourceLen];
	}
}

extern "C" __global__
void scaleRepeatedPow2(float * dest, float * source, int destLen, int srcMask) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid < destLen) {
		dest[tid] *= source[tid & srcMask];
	}
}

extern "C" __global__
void addScaler(float s, float * dest, int destLen) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid < destLen) {
		dest[tid] += s;
	}
}

extern "C" __global__
void setScaler(float s, float * dest, int destLen) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid < destLen) {
		dest[tid] = s;
	}
}

extern "C" __global__
void addChunks(float * dest, float * source, int destLen, int chunkSize) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid < destLen) {
		dest[tid] += source[tid / chunkSize];
	}
}

extern "C" __global__
void subChunks(float * dest, float * source, int destLen, int chunkSize) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid < destLen) {
		dest[tid] -= source[tid / chunkSize];
	}
}

extern "C" __global__
void lessThan(float s, float * v, int n) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid < n) {
    if (v[tid] < s) {
      v[tid] = 1;
    } else {
      v[tid] = 0;
    }
	}
}

extern "C" __global__
void greaterThan(float s, float * v, int n) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid < n) {
    if (v[tid] > s) {
      v[tid] = 1;
    } else {
      v[tid] = 0;
    }
	}
}

extern "C" __global__
void equalTo(float s, float * v, int n) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid < n) {
    if (v[tid] == s) {
      v[tid] = 1;
    } else {
      v[tid] = 0;
    }
	}
}

extern "C" __device__
float addLogPair(float x, float y) {
  float m = max(x, y);
  return logf(expf(x-m) + expf(y-m)) + m;
}

extern "C" __global__
void addLogs(float * dst, float * src, int rowSize) {
  extern __shared__ float chunk[];

  int rowIdx = blockIdx.y * blockDim.x + threadIdx.x;
  if (rowIdx < rowSize) {
    chunk[threadIdx.x] = src[rowIdx+rowSize*blockIdx.x];
  }
  __syncthreads();

  for (int stride = (blockDim.x>>1); stride >= 1; stride >>= 1) {
    if (threadIdx.x < stride && rowIdx+stride < rowSize) {
      chunk[threadIdx.x] = addLogPair(chunk[threadIdx.x],
        chunk[threadIdx.x+stride]);
    }
    __syncthreads();
  }

  if (threadIdx.x == 0) {
    dst[blockIdx.y + blockIdx.x*gridDim.y] = chunk[0];
  }
}

extern "C" __global__
void powScaler(float s, float * dest, int destLen) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid < destLen) {
		dest[tid] = powf(dest[tid], s);
	}
}

extern "C" __global__
void mapForward(float * dst, float * src, int * table, int tableSize) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid < tableSize) {
		dst[tid] = src[table[tid]];
	}
}

extern "C" __global__
void mapBackward(float * dst, float * src, int * table, int tableSize) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid < tableSize) {
		atomicAdd(&dst[table[tid]], src[tid]);
	}
}

extern "C" __global__
void mapMax(int * table, float * data, int rows, int cols) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid < rows) {
		int base = tid * cols;
		float * row = &data[base];
		int maxIdx = 0;
		float maxVal = row[0];
		for (int i = 1; i < cols; ++i) {
			if (row[i] > maxVal) {
				maxVal = row[i];
				maxIdx = i;
			}
		}
		table[tid] = maxIdx + base;
	}
}
